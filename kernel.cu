#include "hip/hip_runtime.h"
// This Program is Written by Abubakr Shafique (abubakr.shafique@gmail.com)
#include "hip/hip_runtime.h"
#include ""
#include "Inversion_CUDA.h"

__global__ void Inversion_CUDA(unsigned char* Image, int Channels);

void Image_Inversion_CUDA(unsigned char* Input_Image, int Height, int Width, int Channels){
	unsigned char* Dev_Input_Image = NULL;

	//allocate the memory in gpu
	hipMalloc((void**)&Dev_Input_Image, Height * Width * Channels);

	//copy data from CPU to GPU
	hipMemcpy(Dev_Input_Image, Input_Image, Height * Width * Channels, hipMemcpyHostToDevice);

	dim3 Grid_Image(Width, Height);
	Inversion_CUDA << <Grid_Image, 1 >> >(Dev_Input_Image, Channels);

	//copy processed data back to cpu from gpu
	hipMemcpy(Input_Image, Dev_Input_Image, Height * Width * Channels, hipMemcpyDeviceToHost);

	//free gpu mempry
	hipFree(Dev_Input_Image);
}

__global__ void Inversion_CUDA(unsigned char* Image, int Channels){
	int x = blockIdx.x;
	int y = blockIdx.y;
	int idx = (x + y * gridDim.x) * Channels;

	for (int i = 0; i < Channels; i++){
		Image[idx + i] = 255 - Image[idx + i];
	}
}